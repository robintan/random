#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define TPB 32
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

void runFloyd(int *mat, const size_t N);
void GenMatrix(int *mat, const size_t N);
void ST_APSP(int *mat, const size_t N);
void printMatrix(int *mat, const size_t N);
bool CmpArray(const int *l, const int *r, const size_t eleNum);

/*
	Generate Matrix
*/
void GenMatrix(int *mat, const size_t N)
{
	for(int i = 0; i < N; i ++)
		for(int j = 0; j < N; j++)
			mat[i*N+j] = (i==j)?0:rand()%32 - 1;
}

/*
	Sequential (Single Thread) APSP on CPU.
*/
void ST_APSP(int *mat, const size_t N)
{
	for(int k = 0; k < N; k ++)
		for(int i = 0; i < N; i ++)
			for(int j = 0; j < N; j ++)
			{
				int i0 = i*N + j;
				int i1 = i*N + k;
				int i2 = k*N + j;
				if(mat[i1] != -1 && mat[i2] != -1)
					mat[i0] = min(mat[i0], mat[i1] + mat[i2]);
			}
}

/*
	Compare two array
*/
bool CmpArray(const int *l, const int *r, const size_t eleNum)
{
	for(int i = 0; i < eleNum; i ++)
		if(l[i] != r[i])
		{
			printf("ERROR: l[%d] = %d, r[%d] = %d\n", i, l[i], i, r[i]);
			return false;
		}
	return true;
}

void printMatrix (int*mat, const size_t N) {
	for (int i = 0; i< N; i++) {
		for(int j= 0; j< N; j++) {
 			int value = mat[i*N +j];
				printf("%d, ",value);
		}
		printf("\n");
	}
}

__global__ void
transpose(int* mat, int* result, const size_t N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < N && j < N)
		result[j*N + i] = mat[i*N + j];
}

/*
	GPU kernel function
*/
__global__ void
floydKernel(int k, int *result_d, const size_t N)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < N && y < N){
		int xk = x*N+k;
		int ky = k*N+y;
		int xy = x*N+y;
		if((result_d[xk] != -1) && (result_d[ky]!=-1))
			result_d[xy] = min(result_d[xy],result_d[xk] + result_d[ky]);
	}
	__syncthreads();
}

/*
	Call kernel function from Host
*/
void runFloyd(int *result, const size_t N)
{
	int size = N * N * sizeof(int);
	int *result_d;
	
	hipMalloc((int **) &result_d, size);
	hipMemcpy(result_d, result, size, hipMemcpyHostToDevice);

	dim3 Grid(N/TPB,N/TPB,1);
	dim3 Block(TPB,TPB,1);
	
	if (N%TPB!=0) {	//ceiling function
		Grid.x++; Grid.y++;
	}
	
	for(int k = 0; k < N; k++){
		floydKernel<<<Grid, Block>>>(k, result_d,N);
	}
	hipMemcpy(result, result_d, size, hipMemcpyDeviceToHost);
	hipFree(result_d);
}

__global__ void
coalesceKernel (int k, int *mat, int *transposed_mat, const size_t N) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < N && y < N) 
		if (transposed_mat[k*N + x] != -1 && mat[k*N + y] != -1) {	//transposed_mat[k,x] = mat[x,k] --> coalesced
			mat[x*N + y] = min(transposed_mat[k*N + x] + mat[k*N + y],mat[x*N+y]);
			transposed_mat[y*N + x] = mat[x*N + y];	//update both to avoid overhead
		}
	__syncthreads();
}

void runFloyd_coalescing(int *result, const size_t N)
{
	int size = N * N * sizeof(int);
	int *result_d;
	hipMalloc((int **) &result_d, size);
	hipMemcpy(result_d, result, size, hipMemcpyHostToDevice);
	
	int *transposed_mat_d;
	hipMalloc((int **) &transposed_mat_d, size);
	
	dim3 Grid(N/TPB,N/TPB,1);
	dim3 Block(TPB,TPB,1);
	if (N%TPB!=0) {	//ceiling function
		Grid.x++; Grid.y++;
	}
	transpose<<<Grid, Block>>>(result_d, transposed_mat_d, N);
	
	for(int k = 0; k < N; ++k) {
		coalesceKernel<<<Grid, Block>>>(k, result_d, transposed_mat_d, N);
	}
	hipMemcpy(result,result_d,size,hipMemcpyDeviceToHost);
	hipFree(transposed_mat_d);
	hipFree(result_d);
}

__global__ void
sharedKernel (int k, int *mat, const size_t N) {
	extern __shared__ int smem[];

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < N && y < N) {
		if (threadIdx.x == 0) {		//to avoid reading from memory all the time, 
									//the number of times it reads from memory is just 2*TPB times.
			smem[TPB+threadIdx.y] = mat[k*N + y];	//KJ
		} if (threadIdx.y == 0) {
			smem[threadIdx.x] = mat[x*N + k];		//IK
		}
		__syncthreads();			//make sure the arrays are filled
		if (smem[threadIdx.x] != -1 && smem[TPB+threadIdx.y] != -1)		//number of times the shared memory is accessed = TPB*TPB
			mat[x*N+y] = min(smem[threadIdx.x] + smem[TPB+threadIdx.y], mat[x*N+y]);
	}
	__syncthreads();
}

void runFloyd_shared(int *result, const size_t N)
{
	int size = N * N * sizeof(int);
	int *result_d;
	hipMalloc((int **) &result_d, size);
	hipMemcpy(result_d, result, size, hipMemcpyHostToDevice);
	
	dim3 Grid(N/TPB,N/TPB,1);
	dim3 Block(TPB,TPB,1);
	if (N%TPB!=0) {	//ceiling function
		Grid.x++; Grid.y++;
	}

	for(int k = 0; k < N; ++k) {
		sharedKernel<<<Grid, Block, (2*TPB)*sizeof(int)>>>(k, result_d, N);
	}
	hipMemcpy(result,result_d,size,hipMemcpyDeviceToHost);
	hipFree(result_d);
}

__global__ void
sharedCoalescedKernel (int k, int *mat, int *transposed_mat, const size_t N) {
	extern __shared__ int smem[];

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < N && y < N) {
		if (threadIdx.x == 0) {		//to avoid reading from memory all the time, 
									//the number of times it reads from memory is just 2*TPB times.
			smem[TPB+threadIdx.y] = mat[k*N + y];			//KJ
		} if (threadIdx.y == 0) {
			smem[threadIdx.x] = transposed_mat[k*N + x];	//KI ==> equivalent to IK of mat
		}
		__syncthreads();			//make sure the arrays are filled
		if (smem[threadIdx.x] != -1 && smem[TPB+threadIdx.y] != -1) {
			mat[x*N+y] = min(smem[threadIdx.x] + smem[TPB+threadIdx.y], mat[x*N+y]);	//number of times being accessed TPB*TPB
			transposed_mat[y*N+x] = mat[x*N+y];
		}
	}
	__syncthreads();
}

void runFloyd_sharedCoalesced(int *result, const size_t N)
{
	int size = N * N * sizeof(int);
	int *result_d;
	hipMalloc((int **) &result_d, size);
	hipMemcpy(result_d, result, size, hipMemcpyHostToDevice);
	
	int *transposed_mat_d;
	hipMalloc((int **) &transposed_mat_d, size);
	
	dim3 Grid(N/TPB,N/TPB,1);
	dim3 Block(TPB,TPB,1);
	if (N%TPB!=0) {	//ceiling function
		Grid.x++; Grid.y++;
	}

	transpose<<<Grid, Block>>>(result_d, transposed_mat_d, N);
	
	for(int k = 0; k < N; ++k) {
		sharedCoalescedKernel<<<Grid, Block, (2*TPB)*sizeof(int)>>>(k, result_d, transposed_mat_d, N);
	}
	hipMemcpy(result,result_d,size,hipMemcpyDeviceToHost);
	hipFree(transposed_mat_d);
	hipFree(result_d);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
	hipEvent_t begin, stop;
	hipEventCreate(&begin);
	hipEventCreate(&stop);
	float dt_ms;
	long int usec;	
	struct timeval start, end;
	
	if (argc < 1) {
		printf("Usage: N [TPB]\n");
		return 0;
	}
	
	// generate a random matrix.
	size_t N = atoi(argv[1]);
	int *mat = (int*)malloc(sizeof(int) * N * N);
	GenMatrix(mat, N);
	
 	// compute the reference result.
	int *ref = (int*)malloc(sizeof(int) * N * N);
	memcpy(ref, mat, sizeof(int) * N * N);
	
	// PERFORM COMPUTATION ON HOST CPU
	gettimeofday(&start,0);
	ST_APSP(ref, N);
	gettimeofday(&end,0);
        printf("Sequential execution time = %ld usecs \n\n", (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec));
    
	// PERFORM COMPUTATION ON GPU
	int *result = (int*)malloc(sizeof(int) * N * N);
	memcpy(result, mat, sizeof(int)*N*N);
	hipEventRecord(begin,0);
  	runFloyd(result, N);
	hipEventRecord(stop,0);

	hipEventSynchronize(begin);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&dt_ms, begin, stop);
	usec = dt_ms *1000;
	printf("CUDA Normal execution time = % ld usecs \n",usec);

	// compare your result with reference result
	if(CmpArray(result, ref, N * N))	printf("Your result is correct.\n\n");
	else								printf("Your result is wrong.\n\n");
	

	// PERFORM COMPUTATION ON GPU WITH MEMORY COALESCING METHOD
	int *coalesced_result = (int*)malloc(sizeof(int) * N * N);
	memcpy(coalesced_result, mat, sizeof(int)*N*N);
	hipEventRecord(begin,0);
	runFloyd_coalescing(coalesced_result, N);
	hipEventRecord(stop,0);
	hipEventSynchronize(begin);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&dt_ms, begin, stop);
	usec = dt_ms *1000;
	printf("CUDA Coalescing execution time = % ld usecs \n",usec);

	if(CmpArray(coalesced_result, ref, N * N))	printf("Your result is correct.\n\n");
	else								printf("Your result is wrong.\n\n");
	
	// PERFORM COMPUTATION ON GPU WITH MEMORY TILING SHARED MEMORY METHOD
	int *shared_result = (int*)malloc(sizeof(int) * N * N);
	memcpy(shared_result, mat, sizeof(int)*N*N);
	hipEventRecord(begin,0);
	runFloyd_shared(shared_result, N);
	hipEventRecord(stop,0);
	hipEventSynchronize(begin);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&dt_ms, begin, stop);
	usec = dt_ms *1000;
	printf("CUDA SM execution time = % ld usecs \n",usec);

	if(CmpArray(shared_result, ref, N * N))	printf("Your result is correct.\n\n");
	else								printf("Your result is wrong.\n\n");
	
	// PERFORM COMPUTATION ON GPU WITH MEMORY TILING SHARED MEMORY AND COALESCING METHOD METHOD
	int *shared_coalesced_result = (int*)malloc(sizeof(int) * N * N);
	memcpy(shared_coalesced_result, mat, sizeof(int)*N*N);
	hipEventRecord(begin,0);
	runFloyd_sharedCoalesced(shared_coalesced_result, N);
	hipEventRecord(stop,0);

	hipEventSynchronize(begin);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&dt_ms, begin, stop);
	usec = dt_ms *1000;
	printf("CUDA SM+Coalesceing execution time = % ld usecs \n",usec);
	
	if(CmpArray(shared_coalesced_result, ref, N * N))	printf("Your result is correct.\n\n");
	else												printf("Your result is wrong.\n\n");

	hipEventDestroy(begin);
	hipEventDestroy(stop);
}




